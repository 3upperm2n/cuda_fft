#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <assert.h>
#include <time.h>	// srand
//#include <math.h>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>



int main(int argc, char **argv)
{
	int fftlen = 1024;

	if(argc == 2) {
		fftlen = atoi(argv[1]);
	}

	assert(argc <= 2 && "Wrong input. Just specify the fft len!");

	printf("Running fft length %d on GPU.\n", fftlen);

	//------------------------------------------------------------------------//
	// host memory
	//------------------------------------------------------------------------//
	hipfftComplex *h_sig;
	hipHostMalloc((void **) &h_sig, sizeof(hipfftComplex) * fftlen);

	srand(time(NULL)); // initialize random seed
	for (int i = 0; i < fftlen; i++) {
		h_sig[i].x = (float)rand() / RAND_MAX;
		h_sig[i].y = 0.0;
	}

	//------------------------------------------------------------------------//
	// device memory
	//------------------------------------------------------------------------//
	hipfftComplex *d_sig, *d_result;
	hipMalloc((void **) &d_sig,    sizeof(hipfftComplex) * fftlen);
	hipMalloc((void **) &d_result, sizeof(hipfftComplex) * fftlen);

	//------------------------------------------------------------------------//
	// copy data from host to device 
	//------------------------------------------------------------------------//
	hipMemcpyAsync(d_sig, h_sig, sizeof(hipfftComplex) * fftlen, hipMemcpyHostToDevice);

	//------------------------------------------------------------------------//
	// set up cuda fft env 
	//------------------------------------------------------------------------//
	hipfftHandle cufft_plan;
	hipfftPlan1d(&cufft_plan, fftlen, HIPFFT_C2C, 1); // batch = 1

	//------------------------------------------------------------------------//
	// gpu timer 
	//------------------------------------------------------------------------//
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int total_iters = 100;
	//------------------------------------------------------------------------//
	// run fft on gpu 
	//------------------------------------------------------------------------//
	hipEventRecord(start, 0);

	for(int i = 0; i<total_iters; i++)
		hipfftExecC2C(cufft_plan, d_sig, d_result, HIPFFT_FORWARD);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float gputime_ms;
	hipEventElapsedTime(&gputime_ms, start, stop);

	float avg_time_ms = gputime_ms / float(total_iters);

	printf("Avg %d fft on GPU : %f ms\n", fftlen, avg_time_ms);

	//------------------------------------------------------------------------//
	// free 
	//------------------------------------------------------------------------//
	checkCudaErrors(hipfftDestroy(cufft_plan)); 	// cuda fft context

	checkCudaErrors(hipHostFree(h_sig));
	checkCudaErrors(hipFree(d_sig));
	checkCudaErrors(hipFree(d_result));
}
