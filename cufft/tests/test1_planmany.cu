#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>

#define DATASIZE 8
#define BATCH 2

/********************/
/* CUDA ERROR CHECK */
/********************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

/********/
/* MAIN */
/********/
int main (int argc, char* argv[])
{
	if(argc != 2) {
		printf("Please specify 1 (plan1d) or 2 (planmany).\n");	
		return -1;
	};

	// --- Host side input data allocation and initialization
	hipfftReal *hostInputData = (hipfftReal*)malloc(DATASIZE*BATCH*sizeof(hipfftReal));
	for (int i=0; i<BATCH; i++) {
		for (int j=0; j<DATASIZE; j++){
			//hostInputData[i*DATASIZE + j] = (cufftReal)(j + 1);
			//hostInputData[i*DATASIZE + j] = 100.f; 
			hostInputData[i*DATASIZE + j] = (float)(j % 10); 
			printf("%f ", hostInputData[i*DATASIZE + j]);
		}
		printf("\n");
	}
	printf("\n");

	// --- Device side input data allocation and initialization
	hipfftReal *deviceInputData;
	gpuErrchk(hipMalloc((void**)&deviceInputData, DATASIZE * BATCH * sizeof(hipfftReal)));
	hipMemcpy(deviceInputData, hostInputData, DATASIZE * BATCH * sizeof(hipfftReal), hipMemcpyHostToDevice);

	// --- Host side output data allocation
	hipfftComplex *hostOutputData = (hipfftComplex*)malloc((DATASIZE / 2 + 1) * BATCH * sizeof(hipfftComplex));

	// --- Device side output data allocation
	hipfftComplex *deviceOutputData; gpuErrchk(hipMalloc((void**)&deviceOutputData, (DATASIZE / 2 + 1) * BATCH * sizeof(hipfftComplex)));

	// --- Batched 1D FFTs
	hipfftHandle handle;
	int rank = 1;                           // --- 1D FFTs
	int n[] = { DATASIZE };                 // --- Size of the Fourier transform
	int istride = 1, ostride = 1;           // --- Distance between two successive input/output elements
	int idist = DATASIZE, odist = (DATASIZE / 2 + 1); // --- Distance between batches
	int inembed[] = { 0 };                  // --- Input size with pitch (ignored for 1D transforms)
	int onembed[] = { 0 };                  // --- Output size with pitch (ignored for 1D transforms)
	int batch = BATCH;                      // --- Number of batched executions


	int plantype = atoi(argv[1]);

	if(plantype == 1) {
		printf("cufftplan1d\n");
		hipfftPlan1d(&handle, DATASIZE, HIPFFT_R2C, BATCH);
	}

	if(plantype == 2) {
		printf("cufftplan2d\n");
		hipfftPlanMany(&handle, rank, n, 
				inembed, istride, idist,
				onembed, ostride, odist, HIPFFT_R2C, batch);
	}


	hipfftExecR2C(handle,  deviceInputData, deviceOutputData);

	// --- Device->Host copy of the results
	gpuErrchk(hipMemcpy(hostOutputData, deviceOutputData, (DATASIZE / 2 + 1) * BATCH * sizeof(hipfftComplex), hipMemcpyDeviceToHost));

	for (int i=0; i<BATCH; i++)
		for (int j=0; j<(DATASIZE / 2 + 1); j++)
			printf("%i %i %f %f\n", i, j, hostOutputData[i*(DATASIZE / 2 + 1) + j].x, hostOutputData[i*(DATASIZE / 2 + 1) + j].y);

	hipfftDestroy(handle);
	gpuErrchk(hipFree(deviceOutputData));
	gpuErrchk(hipFree(deviceInputData));

}
