#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <assert.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include "common.h"

extern const int SigLen;
extern const int FFTRun;
extern const int Trials; 


int main(int argc, char **argv)
{
	int sig_len = SigLen;
	int fft_run = FFTRun;
	int trials  = Trials;

	//-------------------//
	// read cmd options
	//-------------------//
	int i = 0;
	while(i < argc)
	{
		if(argv[i][0]=='-') 
		{
			if(argv[i][1]=='-'){
				// read long options
				if(moreopt(argv[i]))
					fprintf(stderr,"unknown verbose option : %s\n", argv[i]);
			}	

			// read short options
			switch(argv[i][1])
			{
				case 'u':
					usage(argv[0]);
					exit(EXIT_FAILURE);

				case 'l':
					i=read_opt(argc, argv, i, &sig_len, "int");
					break;

				case 'r':
					i=read_opt(argc, argv, i, &fft_run, "int");
					break;

				case 't':
					i=read_opt(argc, argv, i, &trials, "int");
					break;
			}
		}
		i++;
	}

	printf("[LOG] Signal Length: %d\n", sig_len);
	printf("[LOG] FFT run: %d\n", fft_run);
	printf("[LOG] Trials: %d\n", trials);


	printf("[LOG] Start 2d-fft on GPU.\n");

	//------------------------------------------------------------------------//
	// host memory
	//------------------------------------------------------------------------//
	hipfftComplex *h_sig;
	hipHostMalloc((void **) &h_sig, sizeof(hipfftComplex) * sig_len * sig_len);

	srand(time(NULL));
	for (int i = 0; i < sig_len * sig_len; i++) {
			h_sig[i].x = (float)rand() / RAND_MAX;
			h_sig[i].y = (float)rand() / RAND_MAX;
	}

	//------------------------------------------------------------------------//
	// device memory
	//------------------------------------------------------------------------//
	hipfftComplex *d_sig, *d_result;
	hipMalloc((void **) &d_sig,    sizeof(hipfftComplex) * sig_len * sig_len);
	hipMalloc((void **) &d_result, sizeof(hipfftComplex) * sig_len * sig_len);

	//------------------------------------------------------------------------//
	// copy data from host to device 
	//------------------------------------------------------------------------//
	hipMemcpyAsync(d_sig, h_sig, sizeof(hipfftComplex) * sig_len * sig_len, 
			hipMemcpyHostToDevice);

	//------------------------------------------------------------------------//
	// set up cuda fft env 
	//------------------------------------------------------------------------//
	printf("[LOG] Create C2C plan for %i x %i on GPU.\n", sig_len, sig_len);

	hipfftHandle cufft_plan2d;

	hipfftPlan2d(&cufft_plan2d, sig_len, sig_len, HIPFFT_C2C);

	//------------------------------------------------------------------------//
	// gpu timer 
	//------------------------------------------------------------------------//
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//------------------------------------------------------------------------//
	// run fft on gpu 
	//------------------------------------------------------------------------//
	double sum_gputime_ms = 0.0;

	printf("[LOG] Benchmarking cufft ...\n");

	for (int i = 0; i < trials; i++) {

		hipEventRecord(start, 0);

		for (int j = 0; j < fft_run; j++) {
			hipfftExecC2C(cufft_plan2d, d_sig, d_result, HIPFFT_FORWARD);
		}

		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);

		float gputime_ms = 0.0;
		hipEventElapsedTime(&gputime_ms, start, stop);

		sum_gputime_ms += (double) gputime_ms;

		printf("%lf ms (%d iters)\n", gputime_ms, fft_run);
	}

	printf("[LOG] Finished!\n");
	printf("[LOG] Average: %lf sec (per %d iters)\n", sum_gputime_ms * 1e-3 / (double)trials, fft_run);

	//------------------------------------------------------------------------//
	// free 
	//------------------------------------------------------------------------//
	checkCudaErrors(hipfftDestroy(cufft_plan2d)); 	// cuda fft context

	checkCudaErrors(hipHostFree(h_sig));
	checkCudaErrors(hipFree(d_sig));
	checkCudaErrors(hipFree(d_result));
}
