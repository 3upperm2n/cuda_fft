#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <assert.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include "common.h"

extern const int SigLen;
extern const int FFTRun;
extern const int Trials; 


int main(int argc, char **argv)
{
	int targetDev = 0;
	hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, targetDev);
	printf("Device name: %s\n", prop.name);

	hipSetDevice(targetDev);

	/*
	int sig_len = SigLen;
	int fft_run = FFTRun;
	int trials  = Trials;

	//-------------------//
	// read cmd options
	//-------------------//
	int i = 0;
	while(i < argc)
	{
		if(argv[i][0]=='-') 
		{
			if(argv[i][1]=='-'){
				// read long options
				if(moreopt(argv[i]))
					fprintf(stderr,"unknown verbose option : %s\n", argv[i]);
			}	

			// read short options
			switch(argv[i][1])
			{
				case 'u':
					usage(argv[0]);
					exit(EXIT_FAILURE);

				case 'l':
					i=read_opt(argc, argv, i, &sig_len, "int");
					break;

				case 'r':
					i=read_opt(argc, argv, i, &fft_run, "int");
					break;

				case 't':
					i=read_opt(argc, argv, i, &trials, "int");
					break;
			}
		}
		i++;
	}

	printf("[LOG] FFT Length: %d\n", sig_len);
	printf("[LOG] Runs: %d\n", fft_run);
	printf("[LOG] Trials: %d\n", trials);

	*/

	printf("[LOG] Start 1d-fft GPU.\n");

	//------------------------------------------------------------------------//
	// host memory
	//------------------------------------------------------------------------//
	float *h1 =  (float*) malloc(sizeof(float) * 2048 * 64);
	float *h2 =  (float*) malloc(sizeof(float) * 4096 * 64);
	float *h3 =  (float*) malloc(sizeof(float) * 4098 * 64);

	for(int i=0;i<64;i++){
		for(int j=0;j<2048;j++){
			h1[i*2048 + j] = (float)(j+1);
		}
	}

	for(int i=0;i<64;i++){
		for(int j=0;j<4096;j++){
			h2[i*4096 + j] = (float)(j+1);
		}
	}

	for(int i=0;i<64;i++){
		for(int j=0;j<4098;j++){
			h3[i*4098 + j] = (float)(j+1);
		}
	}

	//------------------------------------------------------------------------//
	// gpu timer 
	//------------------------------------------------------------------------//
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//------------------------------------------------------------------------//
	// device memory
	//------------------------------------------------------------------------//
	// ToDo: allocate device memory for host data (d_sig) and  for output results (d_result)
	float *d1,*d2,*d3;
	checkCuda( hipMalloc((void**)&d1, sizeof(float) * 2048 * 64) );
	checkCuda( hipMalloc((void**)&d2, sizeof(float) * 4096 * 64) );
	checkCuda( hipMalloc((void**)&d3, sizeof(float) * 4098 * 64) );

	hipfftComplex *d1_complex, *d2_complex, *d3_complex;
	checkCuda( hipMalloc((void**)&d1_complex, sizeof(hipfftComplex) * 2048 * 64) );
	checkCuda( hipMalloc((void**)&d2_complex, sizeof(hipfftComplex) * 4096 * 64) );
	checkCuda( hipMalloc((void**)&d3_complex, sizeof(hipfftComplex) * 4098 * 64) );

	//------------------------------------------------------------------------//
	// copy data from host to device 
	//------------------------------------------------------------------------//
	checkCuda( hipMemcpy(d1, h1, sizeof(float)*64*2048, hipMemcpyHostToDevice) );
	checkCuda( hipMemcpy(d2, h2, sizeof(float)*64*4096, hipMemcpyHostToDevice) );
	checkCuda( hipMemcpy(d3, h3, sizeof(float)*64*4098, hipMemcpyHostToDevice) );

	//------------------------------------------------------------------------//
	// Create FFT plan 
	//------------------------------------------------------------------------//
	printf("[LOG] 2k fft plan.\n");
	hipfftHandle plan2k;
	if (hipfftPlan1d(&plan2k, 2048, HIPFFT_R2C, 64) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error: 2k Plan creation failed");
		return -1;	
	}	

	printf("[LOG] 2k inverse fft plan.\n");
	hipfftHandle plan2ki;
	if (hipfftPlan1d(&plan2ki, 2048, HIPFFT_C2R, 64) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error: ifft 2k Plan creation failed");
		return -10;	
	}

	printf("[LOG] 4k fft plan.\n");
	hipfftHandle plan4k;
	if (hipfftPlan1d(&plan4k, 4096, HIPFFT_R2C, 64) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error: 4k Plan creation failed");
		return -2;	
	}	

	printf("[LOG] 4k inverse fft plan.\n");
	hipfftHandle plan4ki;
	if (hipfftPlan1d(&plan4ki, 4096, HIPFFT_C2R, 64) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error: ifft 4k Plan creation failed");
		return -20;	
	}	

	printf("[LOG] 4k_a fft plan.\n");
	hipfftHandle plan4k_a;
	if (hipfftPlan1d(&plan4k_a, 4096, HIPFFT_R2C, 64) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error: 4k_a Plan creation failed");
		return -3;	
	}	

	printf("[LOG] 4k_a_i fft plan.\n");
	hipfftHandle plan4k_a_i;
	if (hipfftPlan1d(&plan4k_a_i, 4096, HIPFFT_C2R, 64) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error: ifft 4k_a Plan creation failed");
		return -30;	
	}	


	//------------------------------------------------------------------------//
	// run forward FFT
	//------------------------------------------------------------------------//
	float gputime_ms;

	//--------//
	// 2k fft
	//--------//
	printf("[LOG] Test R2C 2048 fft.\n");

	gputime_ms = 0.f;
	hipEventRecord(start, 0);

	for (int i = 0; i < 100; i++) {
		if (hipfftExecR2C(plan2k, (hipfftReal*)d1, (hipfftComplex*)d1_complex) != HIPFFT_SUCCESS){
			fprintf(stderr, "CUFFT error: ExecR2C 2K Forward failed");
			return -4;	
		}
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gputime_ms, start, stop);
	printf("runtime = %lf (ms)\n", gputime_ms * 0.01);

	//--------//
	// 2k ifft
	//--------//
	printf("[ifft] 2048.\n");

	gputime_ms = 0.f;
	hipEventRecord(start, 0);

	for (int i = 0; i < 100; i++) {
		if (hipfftExecC2R(plan2ki, (hipfftComplex*)d1_complex, (hipfftReal*)d1) != HIPFFT_SUCCESS){
			fprintf(stderr, "CUFFT error: ExecR2C 2K Inverset failed");
			return -40;	
		}
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gputime_ms, start, stop);
	printf("runtime = %lf (ms)\n", gputime_ms * 0.01);

	//--------//
	// 4k fft
	//--------//
	printf("[LOG] Test R2C 4096 fft.\n");

	gputime_ms = 0.f;
	hipEventRecord(start, 0);

	for (int i = 0; i < 100; i++) {
		if (hipfftExecR2C(plan4k, (hipfftReal*)d2, (hipfftComplex*)d2_complex) != HIPFFT_SUCCESS){
			fprintf(stderr, "CUFFT error: ExecR2C 4K Forward failed");
			return -5;	
		}
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gputime_ms, start, stop);
	printf("runtime = %lf (ms)\n", gputime_ms * 0.01);

	//--------//
	// 4k ifft
	//--------//
	printf("[ifft] 4096\n");

	gputime_ms = 0.f;
	hipEventRecord(start, 0);

	for (int i = 0; i < 100; i++) {
		if (hipfftExecC2R(plan4ki, (hipfftComplex*)d2_complex, (hipfftReal*)d2) != HIPFFT_SUCCESS){
			fprintf(stderr, "CUFFT error: ExecR2C 4K Inverse failed");
			return -50;	
		}
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gputime_ms, start, stop);
	printf("runtime = %lf (ms)\n", gputime_ms * 0.01);

	//--------//
	// 4k fft on 4098 
	//--------//
	printf("[LOG] Test R2C 4096 fft on 4098.\n");

	gputime_ms = 0.f;
	hipEventRecord(start, 0);

	for (int i = 0; i < 100; i++) {
		if (hipfftExecR2C(plan4k_a, (hipfftReal*)d3, (hipfftComplex*)d3_complex) != HIPFFT_SUCCESS){
			fprintf(stderr, "CUFFT error: ExecR2C 4K_a Forward failed");
			return -6;	
		}
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gputime_ms, start, stop);
	printf("runtime = %lf (ms)\n", gputime_ms * 0.01);

	//--------//
	// 4k ifft on 4098 
	//--------//
	printf("[ifft] 4096 on 4098\n");

	gputime_ms = 0.f;
	hipEventRecord(start, 0);

	for (int i = 0; i < 100; i++) {
		if (hipfftExecC2R(plan4k_a_i, (hipfftComplex*)d3_complex, (hipfftReal*)d3) != HIPFFT_SUCCESS){
			fprintf(stderr, "CUFFT error: ExecR2C 4K_a Inverse failed");
			return -60;	
		}
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gputime_ms, start, stop);
	printf("runtime = %lf (ms)\n", gputime_ms * 0.01);


	//------------------------------------------------------------------------//
	// free 
	//------------------------------------------------------------------------//
	checkCudaErrors(hipEventDestroy(start));
	checkCudaErrors(hipEventDestroy(stop));

	hipfftDestroy(plan2k);
	hipfftDestroy(plan2ki);

	hipfftDestroy(plan4k);
	hipfftDestroy(plan4ki);

	hipfftDestroy(plan4k_a);
	hipfftDestroy(plan4k_a_i);

	hipFree(d1);
	hipFree(d2);
	hipFree(d3);

	hipFree(d1_complex);
	hipFree(d2_complex);
	hipFree(d3_complex);

	free(h1);
	free(h2);
	free(h3);

	return 0;
}
